#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "kmeans.h"
#include "alloc.h"
#include "error.h"

#ifdef __HIPCC__
inline void checkCuda(hipError_t e)
{
    if (e != hipSuccess)
    {
        // hipGetErrorString() isn't always very helpful. Look up the error
        // number in the hipError_t enum in hip/driver_types.h in the CUDA includes
        // directory for a better explanation.
        error("CUDA Error %d: %s\n", e, hipGetErrorString(e));
    }
}

inline void checkLastCudaError()
{
    checkCuda(hipGetLastError());
}
#endif

__device__ int get_tid()
{
    return blockDim.x * blockIdx.x + threadIdx.x; /* TODO: Calculate 1-Dim global ID of a thread */
}

/* square of Euclid distance between two multi-dimensional points */
__host__ __device__ inline static double euclid_dist_2(int numCoords,
                                                       int numObjs,
                                                       int numClusters,
                                                       double *objects,  // [numObjs][numCoords]
                                                       double *clusters, // [numClusters][numCoords]
                                                       int objectId,
                                                       int clusterId)
{
    int i;
    double ans = 0.0, diff;

    for (i = 0; i < numCoords; i++) 
    {
        diff = objects[objectId * numCoords + i] - clusters[i * numClusters + clusterId];
        ans += diff * diff;
    }

    return (ans);
}

__global__ static void find_nearest_cluster(int numCoords,
                                            int numObjs,
                                            int numClusters,
                                            double *objects,        //  [numObjs][numCoords]
                                            double *deviceClusters, //  [numClusters][numCoords]
                                            int *deviceMembership,  //  [numObjs]
                                            double *devdelta)
{

    /* Get the global ID of the thread. */
    int tid = get_tid();

    /* TODO: Maybe something is missing here... should all threads run this? */
    if (tid >= numObjs)
        return;

    if (1)
    {
        int index, i;
        double dist, min_dist;

        /* find the cluster id that has min distance to object */
        index = 0;
        /* TODO: call min_dist = euclid_dist_2(...) with correct objectId/clusterId */
        min_dist = euclid_dist_2(numCoords, numObjs, numClusters, objects, deviceClusters, tid, 0);

        for (i = 1; i < numClusters; i++)
        {
            /* TODO: call dist = euclid_dist_2(...) with correct objectId/clusterId */
            dist = euclid_dist_2(numCoords, numObjs, numClusters, objects, deviceClusters, tid, i);

            /* no need square root */
            if (dist < min_dist)
            { /* find the min and its array index */
                min_dist = dist;
                index = i;
            }
        }

        if (deviceMembership[tid] != index)
        {
            /* TODO: Maybe something is missing here... is this write safe? */
            // (*devdelta) += 1.0;
            atomicAdd(devdelta, 1.0);
        }

        /* assign the deviceMembership to object objectId */
        deviceMembership[tid] = index;
    }
}

//
//  ----------------------------------------
//  DATA LAYOUT
//
//  objects         [numObjs][numCoords]
//  clusters        [numClusters][numCoords]
//  newClusters     [numClusters][numCoords]
//  deviceObjects   [numObjs][numCoords]
//  deviceClusters  [numClusters][numCoords]
//  ----------------------------------------
//
/* return an array of cluster centers of size [numClusters][numCoords]       */
void kmeans_gpu(double *objects,     /* in: [numObjs][numCoords] */
                int numCoords,       /* no. features */
                int numObjs,         /* no. objects */
                int numClusters,     /* no. clusters */
                double threshold,    /* % objects change membership */
                long loop_threshold, /* maximum number of iterations */
                int *membership,     /* out: [numObjs] */
                double *clusters,    /* out: [numClusters][numCoords] */
                int blockSize)
{
    double timing = wtime(), timing_internal, timer_min = 1e42, timer_max = 0;
    // int loop_iterations = 0;
    int i, j, index, loop = 0;
    int *newClusterSize;              /* [numClusters]: no. objects assigned in each
                                         new cluster */
    double delta = 0, *dev_delta_ptr; /* % of objects change their clusters */
    double **newClusters = (double **)calloc_2d(numClusters, numCoords, sizeof(double));

    double *deviceObjects;
    double *deviceClusters;
    int *deviceMembership;

    printf("\n|-------------Naive GPU Kmeans--------------|\n\n");

    /* initialize membership[] */
    for (i = 0; i < numObjs; i++)
        membership[i] = -1;

    /* need to initialize newClusterSize and newClusters[0] to all 0 */
    newClusterSize = (int *)calloc(numClusters, sizeof(int));
    assert(newClusterSize != NULL);

    timing = wtime() - timing;
    printf("t_alloc: %lf ms\n\n", 1000 * timing);
    timing = wtime();

    const unsigned int numThreadsPerClusterBlock = (numObjs > blockSize) ? blockSize : numObjs;
    const unsigned int numClusterBlocks = (numObjs + numThreadsPerClusterBlock - 1) / numThreadsPerClusterBlock; /* TODO: Calculate Grid size, e.g. number of blocks. */
    const unsigned int clusterBlockSharedDataSize = 0;

    checkCuda(hipMalloc(&deviceObjects, numObjs * numCoords * sizeof(double)));
    checkCuda(hipMalloc(&deviceClusters, numClusters * numCoords * sizeof(double)));
    checkCuda(hipMalloc(&deviceMembership, numObjs * sizeof(int)));
    checkCuda(hipMalloc(&dev_delta_ptr, sizeof(double)));

    timing = wtime() - timing;
    printf("t_alloc_gpu: %lf ms\n\n", 1000 * timing);
    timing = wtime();

    checkCuda(hipMemcpy(deviceObjects, objects,
                         numObjs * numCoords * sizeof(double), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(deviceMembership, membership,
                         numObjs * sizeof(int), hipMemcpyHostToDevice));
    timing = wtime() - timing;
    printf("t_get_gpu: %lf ms\n\n", 1000 * timing);
    timing = wtime();

    double cpu_gpu_trans, gpu_cpu_trans, gpu_part, cpu_part;
    double cpu_gpu_trans_total = 0.0, gpu_cpu_trans_total = 0.0, gpu_part_total = 0.0, cpu_part_total = 0.0;

    do
    {
        timing_internal = wtime();

        /* GPU part: calculate new memberships */

        /* TODO: Copy clusters to deviceClusters
        checkCuda(hipMemcpy(...)); */
        cpu_gpu_trans = wtime();
        checkCuda(hipMemcpy(deviceClusters, clusters, numClusters * numCoords * sizeof(double), hipMemcpyHostToDevice));
        checkCuda(hipMemset(dev_delta_ptr, 0, sizeof(double)));
        cpu_gpu_trans = wtime() - cpu_gpu_trans;
        cpu_gpu_trans_total += cpu_gpu_trans;

        gpu_part = wtime();
        // printf("Launching find_nearest_cluster Kernel with grid_size = %d, block_size = %d, shared_mem = %d KB\n", numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize/1000);
        find_nearest_cluster<<<numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize>>>(numCoords, numObjs, numClusters,
                                                                                                          deviceObjects, deviceClusters, deviceMembership, dev_delta_ptr);

        hipDeviceSynchronize();
        checkLastCudaError();

        gpu_part = wtime() - gpu_part;
        gpu_part_total += gpu_part;
        // printf("Kernels complete for itter %d, updating data in CPU\n", loop);

        gpu_cpu_trans = wtime();
        /* TODO: Copy deviceMembership to membership
        checkCuda(hipMemcpy(...)); */
        checkCuda(hipMemcpy(membership, deviceMembership, numObjs * sizeof(int), hipMemcpyDeviceToHost));

        /* TODO: Copy dev_delta_ptr to &delta
        checkCuda(hipMemcpy(...)); */
        checkCuda(hipMemcpy(&delta, dev_delta_ptr, sizeof(double), hipMemcpyDeviceToHost));
        gpu_cpu_trans = wtime() - gpu_cpu_trans;
        gpu_cpu_trans_total += gpu_cpu_trans;

        cpu_part = wtime();
        /* CPU part: Update cluster centers*/

        for (i = 0; i < numObjs; i++)
        {
            /* find the array index of nestest cluster center */
            index = membership[i];

            /* update new cluster centers : sum of objects located within */
            newClusterSize[index]++;
            for (j = 0; j < numCoords; j++)
                newClusters[index][j] += objects[i * numCoords + j];
        }

        /* average the sum and replace old cluster centers with newClusters */
        for (i = 0; i < numClusters; i++)
        {
            for (j = 0; j < numCoords; j++)
            {
                if (newClusterSize[i] > 0)
                    clusters[i * numCoords + j] = newClusters[i][j] / newClusterSize[i];
                newClusters[i][j] = 0.0; /* set back to 0 */
            }
            newClusterSize[i] = 0; /* set back to 0 */
        }

        delta /= numObjs;
        // printf("delta is %f - ", delta);
        loop++;
        // printf("completed loop %d\n", loop);
        cpu_part = wtime() - cpu_part;
        cpu_part_total += cpu_part;
        timing_internal = wtime() - timing_internal;
        if (timing_internal < timer_min)
            timer_min = timing_internal;
        if (timing_internal > timer_max)
            timer_max = timing_internal;
    } while (delta > threshold && loop < loop_threshold);

    timing = wtime() - timing;

    printf("t_cpu_gpu_trans_avg: %lf ms\n\n", cpu_gpu_trans_total * 1000 / loop);
    printf("t_cpu_gpu_trans_total: %lf ms\n\n", cpu_gpu_trans_total * 1000);

    printf("t_gpu_part_avg: %lf ms\n\n", gpu_part_total * 1000 / loop);
    printf("t_gpu_part_total: %lf ms\n\n", gpu_part_total * 1000);

    printf("t_gpu_cpu_trans_avg: %lf ms\n\n", gpu_cpu_trans_total * 1000 / loop);
    printf("t_gpu_cpu_trans_total: %lf ms\n\n", gpu_cpu_trans_total * 1000);

    printf("t_cpu_part_avg: %lf ms\n\n", cpu_part_total * 1000 / loop);
    printf("t_cpu_part_total: %lf ms\n\n", cpu_part_total * 1000);

    printf("nloops = %d  : total = %lf ms\n\t-> t_loop_avg = %lf ms\n\t-> t_loop_min = %lf ms\n\t-> t_loop_max = %lf ms\n\n|-------------------------------------------|\n",
           loop, 1000 * timing, 1000 * timing / loop, 1000 * timer_min, 1000 * timer_max);

    char outfile_name[1024] = {0};
    sprintf(outfile_name, "Execution_logs/silver1-V100_Sz-%lu_Coo-%d_Cl-%d.csv", numObjs * numCoords * sizeof(double) / (1024 * 1024), numCoords, numClusters);
    FILE *fp = fopen(outfile_name, "a+");
    if (!fp)
        error("Filename %s did not open succesfully, no logging performed\n", outfile_name);
    fprintf(fp, "%s,%d,%lf,%lf,%lf\n", "Naive", blockSize, timing / loop, timer_min, timer_max);
    fclose(fp);
    checkCuda(hipFree(deviceObjects));
    checkCuda(hipFree(deviceClusters));
    checkCuda(hipFree(deviceMembership));

    free(newClusters[0]);
    free(newClusters);
    free(newClusterSize);

    return;
}
